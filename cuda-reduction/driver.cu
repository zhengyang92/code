#include "hip/hip_runtime.h"
#include "stdio.h"

__global__ void add (float *A, float *B, float *C) {
    int i = blockIdx.x * 1024 + threadIdx.x;

    C[i] = A[i] + B[i];
}


int main() {

  int N = 1<<30;
  float *x, *y, *z, *d_x, *d_y, *d_z;
  x = (float*)malloc(N*sizeof(float));
  y = (float*)malloc(N*sizeof(float));
  z = (float*)malloc(N*sizeof(float));

  hipMalloc(&d_x, N*sizeof(float));
  hipMalloc(&d_y, N*sizeof(float));
  hipMalloc(&d_z, N*sizeof(float));

  for (int i = 0; i < N; i++) {
    x[i] = 1.0f;
    y[i] = 5.0f;
  }

  hipMemcpy(d_x, x, N*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_y, y, N*sizeof(float), hipMemcpyHostToDevice);

  add<<<(N+1023)/1024, 1024>>>(d_x, d_y, d_z);


  hipMemcpy(z, d_z, N*sizeof(float), hipMemcpyDeviceToHost);

  float maxError = 0.0f;
  for (int i = 0; i < N; i++) {
    maxError = max(maxError, abs(z[i]-5.0f));
  }
  printf("Max error: %f\n", maxError);

  hipFree(d_x);
  hipFree(d_y);
  hipFree(d_z);

  free (x);
  free (y);
  free (z);


}